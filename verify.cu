#include "hip/hip_runtime.h"
#include "macro.h"
#include "helper.h"
#include "verify.h"
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <map>
#include <cstdlib>
#include <cstdio>
#include <cassert>

void* AllocAlphaScale(hipDataType dtype)
{
    void* ptr = nullptr;
    ptr = malloc(Dtype2Size(dtype));
    switch (dtype) {
        case HIP_R_8I:
            *(reinterpret_cast<char*>(ptr)) = 1;
            break;
        case HIP_R_16F:
            *(reinterpret_cast<half*>(ptr)) = 1.f;
            break;
        case HIP_R_32I:
            *(reinterpret_cast<int*>(ptr)) = 1;
            break;
        case HIP_R_32F:
            *(reinterpret_cast<float*>(ptr)) = 1.f;
            break;
        case HIP_R_64F:
            *(reinterpret_cast<double*>(ptr)) = 1.0;
            break;
        default:
            assert(false);
    }
    return ptr;
}

template <typename data_t>
__global__ void InitMatrixKernal(void* dev_ptr, int w, int h, int ld) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    auto ptr = reinterpret_cast<data_t*>(dev_ptr);
    if (x < ld && y < h) {
        ptr[y * ld + x] = (x < w) ? (threadIdx.y * blockDim.x + threadIdx.x) : 0;
    }
}

void InitMatrix(void* ptr, int w, int h, int ld, hipDataType dtype) 
{
    dim3 block(8, 8);
    dim3 grid;
    grid.x = (ld + block.x - 1) / block.x;
    grid.y = ( h + block.y - 1) / block.y;

    switch (dtype) {

        case HIP_R_8I:
            InitMatrixKernal<char><<<grid, block>>>(ptr, w, h, ld);
            break;
        case HIP_R_16F:
            InitMatrixKernal<half><<<grid, block>>>(ptr, w, h, ld);
            break;
        case HIP_R_32F:
            InitMatrixKernal<float><<<grid, block>>>(ptr, w, h, ld);
            break;
        case HIP_R_64F:
            InitMatrixKernal<double><<<grid, block>>>(ptr, w, h, ld);
        case HIP_C_8I:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<char><<<grid, block>>>(ptr, w, h, ld);
            break;
        case HIP_C_32F:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<float><<<grid, block>>>(ptr, w, h, ld);
            break;
        case HIP_C_64F:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<double><<<grid, block>>>(ptr, w, h, ld);
            break;
        default:
            assert(false);
    }
    RUNTIME_API_CALL(hipStreamSynchronize(0));
}

template <typename data_t>
__global__ void NaiveMatrixTransposeKernel(
    int w, int h,
    const void* src_ptr, void* dst_ptr)
{
    auto src = reinterpret_cast<const data_t*>(src_ptr);
    auto dst = reinterpret_cast<data_t*>(dst_ptr);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h) {
        dst[ x * h + y ] = src[ y * w + x ];
    }
}

void NaiveMatrixTranspose(
    int w, int h,
    const void* src, void* dst,
    hipDataType dtype)
{

    dim3 block(8, 8);
    dim3 grid;
    grid.x = (w + block.x - 1) / block.x;
    grid.y = (h + block.y - 1) / block.y;

    switch (dtype) {
        case HIP_R_8I:
            NaiveMatrixTransposeKernel<char><<<grid, block>>>(w, h, src, dst);
            break;
        case HIP_R_16F:
        case HIP_C_8I:
            NaiveMatrixTransposeKernel<half><<<grid, block>>>(w, h, src, dst);
            break;
        case HIP_R_32I:
        case HIP_R_32F:
            NaiveMatrixTransposeKernel<int><<<grid, block>>>(w, h, src, dst);
            break;
        case HIP_R_64F:
        case HIP_C_32F:
            NaiveMatrixTransposeKernel<double><<<grid, block>>>(w, h, src, dst);
            break;
        case HIP_C_64F:
            NaiveMatrixTransposeKernel<double2><<<grid, block>>>(w, h, src, dst);
            break;
        default:
            assert(false);
    }
    RUNTIME_API_CALL(hipStreamSynchronize(0));
}

template <typename src_t, typename acc_t, typename dst_t>
__global__ void NaiveGemmKernelNN(
    int m, int n, int k,
    const void* A_ptr, int lda,
    const void* B_ptr, int ldb,
    void* C_ptr, int ldc) 
{
    auto A = reinterpret_cast<const src_t*>(A_ptr);
    auto B = reinterpret_cast<const src_t*>(B_ptr);
    auto C = reinterpret_cast<dst_t*>(C_ptr);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    acc_t sum = 0;

    if (x < m && y < n) {
        for (int i = 0; i < k; ++i) {
            sum += static_cast<acc_t>(A[i * lda + x]) * static_cast<acc_t>(B[y * ldb + i]);
        }
        C[y * ldc + x] = static_cast<dst_t>(sum);
    }
}

void NaiveGemmNN(
    int m, int n, int k,
    const void* A, int lda,
    const void* B, int ldb,
    void* C, int ldc,
    int gemm_type) 
{

    dim3 block(8, 8);
    dim3 grid;
    grid.x = (m + block.x - 1) / block.x;
    grid.y = (n + block.y - 1) / block.y;
    switch (gemm_type) {
        case 0: // HIP_R_16F, HIP_R_16F, HIP_R_16F, HIP_R_16F
            NaiveGemmKernelNN<half, float, half><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 1: // HIP_R_32I, HIP_R_8I,  HIP_R_8I,  HIP_R_32I
            NaiveGemmKernelNN<char, int, int><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 2: // HIP_R_32F, HIP_R_16F, HIP_R_16F, HIP_R_16F
            NaiveGemmKernelNN<half, float, half><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 3: // HIP_R_32F, HIP_R_8I,  HIP_R_8I,  HIP_R_32F
            NaiveGemmKernelNN<char, float, float><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 4: // HIP_R_32F, HIP_R_16F, HIP_R_16F, HIP_R_32F
            NaiveGemmKernelNN<half, float, float><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 5: // HIP_R_32F, HIP_R_32F, HIP_R_32F, HIP_R_32F
            NaiveGemmKernelNN<float, float, float><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 6: // HIP_R_64F, HIP_R_64F, HIP_R_64F, HIP_R_64F
            NaiveGemmKernelNN<double, double, double><<<grid, block>>>(m, n, k,
                A, lda, B, ldb, C, ldc);
            break;
        case 7: // HIP_C_32F, HIP_C_8I,  HIP_C_8I,  HIP_C_32F
        case 8: // HIP_C_32F, HIP_C_32F, HIP_C_32F, HIP_C_32F
        case 9: // HIP_C_64F, HIP_C_64F, HIP_C_64F, HIP_C_64F
        default:
            assert(false);
    }
    RUNTIME_API_CALL(hipStreamSynchronize(0));
}

int GetGemmTypeId(hipDataType compute_type,
    hipDataType src_type, hipDataType dst_type) 
{
    switch (compute_type) {
        case HIP_R_16F: return 0;
        case HIP_R_32I: return 1;
        case HIP_R_32F:
            switch (src_type) {
                case HIP_R_16F: return (dst_type == HIP_R_16F) ? 2 : 4;
                case HIP_R_8I: return 3;
                case HIP_R_32F: return 5;
                default: assert(false);
            }
        case HIP_R_64F: return 6;
        case HIP_C_32F: return (src_type == HIP_C_8I) ? 7 : 8;
        case HIP_C_64F: return 9;
        default: assert(false);
    }
    return -1;
}

void NaiveGemm(
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m, int n, int k,
    const void* A, hipDataType a_type, int lda,
    const void* B, hipDataType b_type, int ldb,
    void* C, hipDataType c_type, int ldc,
    hipDataType compute_type) 
{
    int src_dtype_size = Dtype2Size(a_type);
    void* dev_A = (void*)A;
    int trans_lda = lda;
    if (transa == HIPBLAS_OP_T) {
        RUNTIME_API_CALL(hipMalloc(&dev_A, m * lda * src_dtype_size));
        NaiveMatrixTranspose(lda, m, A, dev_A, a_type);
        trans_lda = m;
    }

    void* dev_B = (void*)B;
    int trans_ldb = ldb;
    if (transb == HIPBLAS_OP_T) {
        RUNTIME_API_CALL(hipMalloc(&dev_B, k * ldb * src_dtype_size));
        NaiveMatrixTranspose(ldb, k, B, dev_B, b_type);
        trans_ldb = k;
    }

    auto gemm_type = GetGemmTypeId(compute_type, a_type, c_type);
    NaiveGemmNN(m, n, k, dev_A, trans_lda, dev_B, trans_ldb, C, ldc, gemm_type);
    if (dev_A != A) RUNTIME_API_CALL(hipFree(dev_A));
    if (dev_B != B) RUNTIME_API_CALL(hipFree(dev_B));
}

template<typename T>
struct AbsMinus {
    __thrust_exec_check_disable__
    __host__ __device__ T operator()(const T &lhs, const T &rhs) const {
        return (lhs > rhs) ? lhs - rhs : rhs - lhs;
    }
};

template <typename T>
bool VerifyT(const void* x_ptr, const void* y_ptr, int count) {
    auto x = reinterpret_cast<const T*>(x_ptr);
    auto y = reinterpret_cast<const T*>(y_ptr);

    T init = 0;
    thrust::maximum<T> binary_op1;
    AbsMinus<T> binary_op2;

    auto result = thrust::inner_product(thrust::device, 
        x, x + count, y, init, binary_op1, binary_op2);

    if (static_cast<double>(result) > 1e-6) {
        //std::cerr << "error: " << result << std::endl;
        return false;
    }
    else {
        //std::cout << "PASSED" << std::endl;
        return true;
    }
}

std::ostream& operator<<(std::ostream& os, const half& x) {
    os << __half2float(x);
    return os;
}

bool Verify(const void* x, const void* y, int count, hipDataType dtype) {
    switch (dtype) {
        case HIP_R_16F:
            return VerifyT<half>(x, y, count);
        case HIP_R_32I:
            return VerifyT<int>(x, y, count);
        case HIP_R_32F:
            return VerifyT<float>(x, y, count);
        case HIP_R_64F:
            return VerifyT<double>(x, y, count);
        case HIP_C_32F:
            return VerifyT<float>(x, y, 2 * count);
        case HIP_C_64F:
            return VerifyT<double>(x, y, 2 * count);
        default:
            assert(false);
    }
    return false;
}

template <typename data_t>
void PrintMatrixT(const void* ptr, int w, int h, int ld)
{
    auto dev_ptr = reinterpret_cast<const data_t*>(ptr);
    size_t size = ld * h * sizeof(data_t);
    data_t* host_ptr = (data_t*)malloc(size);
    RUNTIME_API_CALL(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost));

    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < ld; ++x) {
            std::cout << +host_ptr[y * ld + x] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << "\n" << std::endl;
    free(host_ptr);
}

template <>
void PrintMatrixT<half>(const void* ptr, int w, int h, int ld)
{
    auto dev_ptr = reinterpret_cast<const half*>(ptr);
    size_t size = ld * h * sizeof(half);
    half* host_ptr = (half*)malloc(size);
    RUNTIME_API_CALL(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost));

    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < ld; ++x) {
            std::cout << host_ptr[y * ld + x] << ", ";
        }
        std::cout << std::endl;
    }
    std::cout << "\n" << std::endl;
    free(host_ptr);
}

void PrintMatrix(const void* dev_ptr, int w, int h,
    int ld, hipDataType dtype)
{
    switch (dtype) {
        case HIP_R_8I:
            PrintMatrixT<char>(dev_ptr, w, h, ld);
            break;
        case HIP_R_16F:
            PrintMatrixT<half>(dev_ptr, w, h, ld);
            break;
        case HIP_R_32I:
            PrintMatrixT<int>(dev_ptr, w, h, ld);
            break;
        case HIP_R_32F:
            PrintMatrixT<float>(dev_ptr, w, h, ld);
            break;
        case HIP_R_64F:
            PrintMatrixT<double>(dev_ptr, w, h, ld);
            break;
        case HIP_C_8I:
            PrintMatrixT<char>(dev_ptr, 2 * w, h, 2 * ld);
            break;
        case HIP_C_32F:
            PrintMatrixT<float>(dev_ptr, 2 * w, h, 2 * ld);
            break;
        case HIP_C_64F:
            PrintMatrixT<double>(dev_ptr, 2 * w, h, 2 * ld);
            break;
        default:
            assert(false);
    }
}
