#include "hip/hip_runtime.h"
#include "macro.h"
#include "helper.h"
#include "verify.h"
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <map>
#include <cstdlib>
#include <cstdio>
#include <cassert>

void* AllocAlphaScale(hipDataType dtype)
{
    void* ptr = nullptr;
    ptr = malloc(Dtype2Size(dtype));
    switch (dtype) {
        case HIP_R_8I:
            *(reinterpret_cast<char*>(ptr)) = 1;
            break;
        case HIP_R_16F:
            *(reinterpret_cast<half*>(ptr)) = 1.f;
            break;
        case HIP_R_32I:
            *(reinterpret_cast<int*>(ptr)) = 1;
            break;
        case HIP_R_32F:
            *(reinterpret_cast<float*>(ptr)) = 1.f;
            break;
        case HIP_R_64F:
            *(reinterpret_cast<double*>(ptr)) = 1.0;
            break;
        default:
            assert(false);
    }
    return ptr;
}

template <typename data_t>
__global__ void InitMatrixKernal(data_t* ptr, int w, int h, int ld) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < ld && y < h) {
        ptr[y * ld + x] = (x < w) ? (threadIdx.y * blockDim.x + threadIdx.x) : 0;
    }
}

void InitMatrix(void* ptr, int w, int h, int ld, hipDataType dtype) 
{
    dim3 block(8, 8);
    dim3 grid;
    grid.x = (ld + block.x - 1) / block.x;
    grid.y = ( h + block.y - 1) / block.y;

    switch (dtype) {

        case HIP_R_8I:
            InitMatrixKernal<char><<<grid, block>>>(reinterpret_cast<char*>(ptr), w, h, ld);
            break;
        case HIP_R_16F:
            InitMatrixKernal<half><<<grid, block>>>(reinterpret_cast<half*>(ptr), w, h, ld);
            break;
        case HIP_R_32F:
            InitMatrixKernal<float><<<grid, block>>>(reinterpret_cast<float*>(ptr), w, h, ld);
            break;
        case HIP_R_64F:
            InitMatrixKernal<double><<<grid, block>>>(reinterpret_cast<double*>(ptr), w, h, ld);
        case HIP_C_8I:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<char><<<grid, block>>>(reinterpret_cast<char*>(ptr), 2 * w, h, 2 * ld);
            break;
        case HIP_C_32F:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<float><<<grid, block>>>(reinterpret_cast<float*>(ptr), 2 * w, h, 2 * ld);
            break;
        case HIP_C_64F:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<double><<<grid, block>>>(reinterpret_cast<double*>(ptr), 2 * w, h, 2 * ld);
            break;
        default:
            assert(false);
    }
    RUNTIME_API_CALL(hipStreamSynchronize(0));
}

template <typename T>
__global__ void NaiveMatrixTransposeKernel(
    int w, int h,
    const T* src, T* dst) 
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h) {
        dst[ x * h + y ] = src[ y * w + x ];
    }
}

void NaiveMatrixTranspose(
    int w, int h,
    void* src, void* dst,
    hipDataType dtype)
{

    dim3 block(8, 8);
    dim3 grid;
    grid.x = (w + block.x - 1) / block.x;
    grid.y = (h + block.y - 1) / block.y;

    switch (dtype) {
        case HIP_R_8I:
            NaiveMatrixTransposeKernel<char><<<grid, block>>>(w, h, reinterpret_cast<char*>(src), reinterpret_cast<char*>(dst));
            break;
        case HIP_R_16F:
        case HIP_C_8I:
            NaiveMatrixTransposeKernel<half><<<grid, block>>>(w, h, reinterpret_cast<half*>(src), reinterpret_cast<half*>(dst));
            break;
        case HIP_R_32I:
        case HIP_R_32F:
            NaiveMatrixTransposeKernel<int><<<grid, block>>>(w, h, reinterpret_cast<int*>(src), reinterpret_cast<int*>(dst));
            break;
        case HIP_R_64F:
        case HIP_C_32F:
            NaiveMatrixTransposeKernel<double><<<grid, block>>>(w, h, reinterpret_cast<double*>(src), reinterpret_cast<double*>(dst));
            break;
        case HIP_C_64F:
            NaiveMatrixTransposeKernel<double2><<<grid, block>>>(w, h, reinterpret_cast<double2*>(src), reinterpret_cast<double2*>(dst));
            break;
        default:
            assert(false);
    }
    RUNTIME_API_CALL(hipStreamSynchronize(0));
}

template <typename src_t, typename acc_t, typename dst_t>
__global__ void NaiveGemmKernelNN(
    int m, int n, int k,
    src_t* A, int lda,
    src_t* B, int ldb,
    dst_t* C, int ldc) 
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    acc_t sum = 0;

    if (x < m && y < n) {
        for (int i = 0; i < k; ++i) {
            sum += static_cast<acc_t>(A[i * lda + x]) * static_cast<acc_t>(B[y * ldb + i]);
        }
        C[y * ldc + x] = static_cast<dst_t>(sum);
    }
}

void NaiveGemmNN(
    int m, int n, int k,
    void* A, int lda,
    void* B, int ldb,
    void* C, int ldc,
    int gemm_type) 
{

    dim3 block(8, 8);
    dim3 grid;
    grid.x = (m + block.x - 1) / block.x;
    grid.y = (n + block.y - 1) / block.y;
    switch (gemm_type) {
        case 0: // HIP_R_16F, HIP_R_16F, HIP_R_16F, HIP_R_16F
            NaiveGemmKernelNN<half, float, half><<<grid, block>>>(m, n, k,
                reinterpret_cast<half*>(A), lda,
                reinterpret_cast<half*>(B), ldb,
                reinterpret_cast<half*>(C), ldc);
            break;
        case 1: // HIP_R_32I, HIP_R_8I,  HIP_R_8I,  HIP_R_32I
            NaiveGemmKernelNN<char, int, int><<<grid, block>>>(m, n, k,
                reinterpret_cast<char*>(A), lda,
                reinterpret_cast<char*>(B), ldb,
                reinterpret_cast<int*>(C), ldc);
            break;
        case 2: // HIP_R_32F, HIP_R_16F, HIP_R_16F, HIP_R_16F
            NaiveGemmKernelNN<half, float, half><<<grid, block>>>(m, n, k,
                reinterpret_cast<half*>(A), lda,
                reinterpret_cast<half*>(B), ldb,
                reinterpret_cast<half*>(C), ldc);
            break;
        case 3: // HIP_R_32F, HIP_R_8I,  HIP_R_8I,  HIP_R_32F
            NaiveGemmKernelNN<char, float, float><<<grid, block>>>(m, n, k,
                reinterpret_cast<char*>(A), lda,
                reinterpret_cast<char*>(B), ldb,
                reinterpret_cast<float*>(C), ldc);
            break;
        case 4: // HIP_R_32F, HIP_R_16F, HIP_R_16F, HIP_R_32F
            NaiveGemmKernelNN<half, float, float><<<grid, block>>>(m, n, k,
                reinterpret_cast<half*>(A), lda,
                reinterpret_cast<half*>(B), ldb,
                reinterpret_cast<float*>(C), ldc);
            break;
        case 5: // HIP_R_32F, HIP_R_32F, HIP_R_32F, HIP_R_32F
            NaiveGemmKernelNN<float, float, float><<<grid, block>>>(m, n, k,
                reinterpret_cast<float*>(A), lda,
                reinterpret_cast<float*>(B), ldb,
                reinterpret_cast<float*>(C), ldc);
            break;
        case 6: // HIP_R_64F, HIP_R_64F, HIP_R_64F, HIP_R_64F
            NaiveGemmKernelNN<double, double, double><<<grid, block>>>(m, n, k,
                reinterpret_cast<double*>(A), lda,
                reinterpret_cast<double*>(B), ldb,
                reinterpret_cast<double*>(C), ldc);
            break;
        case 7: // HIP_C_32F, HIP_C_8I,  HIP_C_8I,  HIP_C_32F
        case 8: // HIP_C_32F, HIP_C_32F, HIP_C_32F, HIP_C_32F
        case 9: // HIP_C_64F, HIP_C_64F, HIP_C_64F, HIP_C_64F
        default:
            assert(false);
    }
    RUNTIME_API_CALL(hipStreamSynchronize(0));
}

int GetGemmTypeId(hipDataType compute_type,
    hipDataType src_type, hipDataType dst_type) 
{
    switch (compute_type) {
        case HIP_R_16F: return 0;
        case HIP_R_32I: return 1;
        case HIP_R_32F:
            switch (src_type) {
                case HIP_R_16F: return (dst_type == HIP_R_16F) ? 2 : 4;
                case HIP_R_8I: return 3;
                case HIP_R_32F: return 5;
                default: assert(false);
            }
        case HIP_R_64F: return 6;
        case HIP_C_32F: return (src_type == HIP_C_8I) ? 7 : 8;
        case HIP_C_64F: return 9;
        default: assert(false);
    }
    return -1;
}

void NaiveGemm(
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m, int n, int k,
    void* A, hipDataType a_type, int lda,
    void* B, hipDataType b_type, int ldb,
    void* C, hipDataType c_type, int ldc,
    hipDataType compute_type) 
{
    int src_dtype_size = Dtype2Size(a_type);
    void* dev_A = A;
    int trans_lda = lda;
    if (transa == HIPBLAS_OP_T) {
        RUNTIME_API_CALL(hipMalloc(&dev_A, m * lda * src_dtype_size));
        NaiveMatrixTranspose(lda, m, A, dev_A, a_type);
        trans_lda = m;
    }

    void* dev_B = B;
    int trans_ldb = ldb;
    if (transb == HIPBLAS_OP_T) {
        RUNTIME_API_CALL(hipMalloc(&dev_B, k * ldb * src_dtype_size));
        NaiveMatrixTranspose(ldb, k, B, dev_B, b_type);
        trans_ldb = k;
    }

    auto gemm_type = GetGemmTypeId(compute_type, a_type, c_type);
    NaiveGemmNN(m, n, k, dev_A, trans_lda, dev_B, trans_ldb, C, ldc, gemm_type);
    if (dev_A != A) RUNTIME_API_CALL(hipFree(dev_A));
    if (dev_B != B) RUNTIME_API_CALL(hipFree(dev_B));
}

template<typename T>
struct AbsMinus {
    __thrust_exec_check_disable__
    __host__ __device__ T operator()(const T &lhs, const T &rhs) const {
        return (lhs > rhs) ? lhs - rhs : rhs - lhs;
    }
};

template <typename T>
bool VerifyT(T* x, T* y, int count) {
    T init = 0;
    thrust::maximum<T> binary_op1;
    AbsMinus<T> binary_op2;

    auto result = thrust::inner_product(thrust::device, 
        x, x + count, y, init, binary_op1, binary_op2);

    if (static_cast<double>(result) > 1e-6) {
        //std::cerr << "error: " << result << std::endl;
        return false;
    }
    else {
        //std::cout << "PASSED" << std::endl;
        return true;
    }
}

std::ostream& operator<<(std::ostream& os, const half& x) {
    os << __half2float(x);
    return os;
}

bool Verify(void* x, void* y, int count, hipDataType dtype) {
    switch (dtype) {
        case HIP_R_16F:
            return VerifyT<half>(reinterpret_cast<half*>(x), reinterpret_cast<half*>(y), count);
        case HIP_R_32I:
            return VerifyT<int>(reinterpret_cast<int*>(x), reinterpret_cast<int*>(y), count);
        case HIP_R_32F:
            return VerifyT<float>(reinterpret_cast<float*>(x), reinterpret_cast<float*>(y), count);
        case HIP_R_64F:
            return VerifyT<double>(reinterpret_cast<double*>(x), reinterpret_cast<double*>(y), count);
        case HIP_C_32F:
            return VerifyT<float>(reinterpret_cast<float*>(x), reinterpret_cast<float*>(y), 2 * count);
        case HIP_C_64F:
            return VerifyT<double>(reinterpret_cast<double*>(x), reinterpret_cast<double*>(y), 2 * count);
        default:
            assert(false);
    }
    return false;
}

void PrintMatrix(float* dev_ptr, int w, int h, int ld)
{
    size_t size = ld * h * sizeof(float);
    float* host_ptr = (float*)malloc(size);
    RUNTIME_API_CALL(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost));
    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < ld; ++x) {
            printf("%.f, ", host_ptr[y * ld + x]);
        }
        printf("\n");
    }
    printf("\n\n");
    free(host_ptr);
}