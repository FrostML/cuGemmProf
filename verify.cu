#include "hip/hip_runtime.h"
#include "macro.h"
#include "verify.h"
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cassert>

void* AllocAlphaScale(hipDataType dtype) {
    void* ptr = nullptr;
    switch (dtype) {

        case HIP_R_8I:
            ptr = malloc(1);
            *(reinterpret_cast<char*>(ptr)) = 1;
            break;
        case HIP_R_16F:
            ptr = malloc(2);
            *(reinterpret_cast<__half*>(ptr)) = __float2half(1.f);
            break;
        case HIP_R_32I:
            ptr = malloc(4);
            *(reinterpret_cast<int*>(ptr)) = 1;
            break;
        case HIP_R_32F:
            ptr = malloc(4);
            *(reinterpret_cast<float*>(ptr)) = 1.f;
            break;
        case HIP_R_64F:
            ptr = malloc(8);
            *(reinterpret_cast<double*>(ptr)) = 1.0;
            break;
        default:
            assert(false);
    }
    return ptr;
}

template <typename data_t>
__global__ void InitMatrixKernal(data_t* ptr, int w, int h, int ld) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < ld && y < h) {
        ptr[y * ld + x] = (x < w) ? static_cast<data_t>(threadIdx.y * blockDim.x + threadIdx.x) : 0;
    }
}

void InitMatrix(void* ptr, int w, int h, int ld, hipDataType dtype) {

    dim3 block(8, 8);
    dim3 grid;
    grid.x = (ld + block.x - 1) / block.x;
    grid.y = ( h + block.y - 1) / block.y;

    switch (dtype) {

        case HIP_R_8I:
            InitMatrixKernal<char><<<grid, block>>>(reinterpret_cast<char*>(ptr), w, h, ld);
            break;
        case HIP_R_16F:
            InitMatrixKernal<__half><<<grid, block>>>(reinterpret_cast<__half*>(ptr), w, h, ld);
            break;
        case HIP_R_32F:
            InitMatrixKernal<float><<<grid, block>>>(reinterpret_cast<float*>(ptr), w, h, ld);
            break;
        case HIP_R_64F:
            InitMatrixKernal<double><<<grid, block>>>(reinterpret_cast<double*>(ptr), w, h, ld);
        case HIP_C_8I:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<char><<<grid, block>>>(reinterpret_cast<char*>(ptr), 2 * w, h, 2 * ld);
            break;
        case HIP_C_32F:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<float><<<grid, block>>>(reinterpret_cast<float*>(ptr), 2 * w, h, 2 * ld);
            break;
        case HIP_C_64F:
            grid.x = (2 * ld + block.x - 1) / block.x;
            InitMatrixKernal<double><<<grid, block>>>(reinterpret_cast<double*>(ptr), 2 * w, h, 2 * ld);
            break;
        default:
            assert(false);
    }
    RUNTIME_API_CALL(hipStreamSynchronize(0));
}

template <typename src_t, typename acc_t, typename dst_t>
__global__ void NaiveGemmKernelNN(
    int m, int n, int k,
    src_t* A, int lda,
    src_t* B, int ldb,
    dst_t* C, int ldc) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    acc_t sum = 0;

    if (x < m && y < n) {
        for (int i = 0; i < k; ++i) {
            sum += static_cast<acc_t>(A[i * lda + x]) * static_cast<acc_t>(B[y * ldb + i]);
        }
        C[y * ldc + x] = static_cast<dst_t>(sum);
    }
}

void NaiveGemmNN(
    int m, int n, int k,
    void* A, int lda,
    void* B, int ldb,
    void* C, int ldc,
    int gemm_type) {

    dim3 block(8, 8);
    dim3 grid;
    grid.x = (m + block.x - 1) / block.x;
    grid.y = (n + block.y - 1) / block.y;
    switch (gemm_type) {
        case 0: // HIP_R_16F, HIP_R_16F, HIP_R_16F, HIP_R_16F
            NaiveGemmKernelNN<__half, float, __half><<<grid, block>>>(m, n, k,
                reinterpret_cast<__half*>(A), lda, 
                reinterpret_cast<__half*>(B), ldb, 
                reinterpret_cast<__half*>(C), ldc);
            break;
        case 1: // HIP_R_32I, HIP_R_8I,  HIP_R_8I,  HIP_R_32I
            NaiveGemmKernelNN<char, int, int><<<grid, block>>>(m, n, k,
                reinterpret_cast<char*>(A), lda, 
                reinterpret_cast<char*>(B), ldb, 
                reinterpret_cast<int*>(C), ldc);
            break;
        case 2: // HIP_R_32F, HIP_R_16F, HIP_R_16F, HIP_R_16F
            NaiveGemmKernelNN<__half, float, __half><<<grid, block>>>(m, n, k,
                reinterpret_cast<__half*>(A), lda, 
                reinterpret_cast<__half*>(B), ldb, 
                reinterpret_cast<__half*>(C), ldc);
            break;
        case 3: // HIP_R_32F, HIP_R_8I,  HIP_R_8I,  HIP_R_32F
            NaiveGemmKernelNN<char, float, float><<<grid, block>>>(m, n, k,
                reinterpret_cast<char*>(A), lda, 
                reinterpret_cast<char*>(B), ldb, 
                reinterpret_cast<float*>(C), ldc);
            break;
        case 4: // HIP_R_32F, HIP_R_16F, HIP_R_16F, HIP_R_32F
            NaiveGemmKernelNN<__half, float, float><<<grid, block>>>(m, n, k,
                reinterpret_cast<__half*>(A), lda, 
                reinterpret_cast<__half*>(B), ldb, 
                reinterpret_cast<float*>(C), ldc);
            break;
        case 5: // HIP_R_32F, HIP_R_32F, HIP_R_32F, HIP_R_32F
            NaiveGemmKernelNN<float, float, float><<<grid, block>>>(m, n, k,
                reinterpret_cast<float*>(A), lda, 
                reinterpret_cast<float*>(B), ldb, 
                reinterpret_cast<float*>(C), ldc);
            break;
        case 6: // HIP_R_64F, HIP_R_64F, HIP_R_64F, HIP_R_64F
            NaiveGemmKernelNN<double, double, double><<<grid, block>>>(m, n, k,
                reinterpret_cast<double*>(A), lda, 
                reinterpret_cast<double*>(B), ldb, 
                reinterpret_cast<double*>(C), ldc);
            break;
        case 7: // HIP_C_32F, HIP_C_8I,  HIP_C_8I,  HIP_C_32F
        case 8: // HIP_C_32F, HIP_C_32F, HIP_C_32F, HIP_C_32F
        case 9: // HIP_C_64F, HIP_C_64F, HIP_C_64F, HIP_C_64F
        default:
            assert(false);
    }
    RUNTIME_API_CALL(hipStreamSynchronize(0));
}

template<typename T>
struct AbsMinus {
    __thrust_exec_check_disable__
    __host__ __device__ T operator()(const T &lhs, const T &rhs) const {
        return (lhs > rhs) ? lhs - rhs : rhs - lhs;
    }
};

template <typename T>
bool VerifyT(T* x, T* y, int count) {
    T init = 0;
    thrust::maximum<T> binary_op1;
    AbsMinus<T> binary_op2;

    auto result = thrust::inner_product(thrust::device, 
        x, x + count, y, init, binary_op1, binary_op2);

    if (static_cast<double>(result) > 1e-6) {
        std::cerr << "error: " << result << std::endl;
        return false;
    }
    else {
        std::cout << "PASSED" << std::endl;
        return true;
    }
}

bool Verify(void* x, void* y, int count, hipDataType dtype) {
    switch (dtype) {
        case HIP_R_16F:
            assert(false);
            //return VerifyT<__half>(reinterpret_cast<__half*>(x), reinterpret_cast<__half*>(y), count);
        case HIP_R_32I:
            return VerifyT<int>(reinterpret_cast<int*>(x), reinterpret_cast<int*>(y), count);
        case HIP_R_32F:
            return VerifyT<float>(reinterpret_cast<float*>(x), reinterpret_cast<float*>(y), count);
        case HIP_R_64F:
            return VerifyT<double>(reinterpret_cast<double*>(x), reinterpret_cast<double*>(y), count);
        case HIP_C_32F:
            return VerifyT<float>(reinterpret_cast<float*>(x), reinterpret_cast<float*>(y), 2 * count);
        case HIP_C_64F:
            return VerifyT<double>(reinterpret_cast<double*>(x), reinterpret_cast<double*>(y), 2 * count);
        default:
            assert(false);
    }
    return false;
}

void PrintMatrix(float* dev_ptr, int w, int h, int ld)
{
    size_t size = ld * h * sizeof(float);
    float* host_ptr = (float*)malloc(size);
    RUNTIME_API_CALL(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost));
    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < ld; ++x) {
            printf("%.f, ", host_ptr[y * ld + x]);
        }
        printf("\n");
    }
    printf("\n\n");
    free(host_ptr);
}